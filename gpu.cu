#include "hip/hip_runtime.h"
// Imports for cuda and reading/writing files
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "lib/kmeans.hpp"

// This the the amount of songs in the data
#define N 42305
// This is the size of the centroids so the amount of k
#define K 12
#define ROUNDS 20

__global__ void findClosestCentroid(song *data, int *cluster_assignment, song *centroids)
{
    // Get id for datapoint to be updated
    const int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Check the bounds
    if (id >= N)
        return;

    // Find the closest centroid to the datapoint
    float minDistance = INFINITY;
    int closest_centroid = -1;

    for (int c = 0; c < K; ++c)
    {
        float total = 0;
        total += (data[id].danceability - centroids[c].danceability) * (data[id].danceability - centroids[c].danceability);
        total += (data[id].energy - centroids[c].energy) * (data[id].energy - centroids[c].energy);
        total += (data[id].loudness - centroids[c].loudness) * (data[id].loudness - centroids[c].loudness);
        total += (data[id].speechiness - centroids[c].speechiness) * (data[id].speechiness - centroids[c].speechiness);
        total += (data[id].acousticness - centroids[c].acousticness) * (data[id].acousticness - centroids[c].acousticness);
        total += (data[id].instrumental - centroids[c].instrumental) * (data[id].instrumental - centroids[c].instrumental);
        total += (data[id].liveness - centroids[c].liveness) * (data[id].liveness - centroids[c].liveness);
        total += (data[id].valence - centroids[c].valence) * (data[id].valence - centroids[c].valence);
        total += (data[id].tempo - centroids[c].tempo) * (data[id].tempo - centroids[c].tempo);
        float dist = sqrtf(total);

        if (dist < minDistance)
        {
            minDistance = dist;
            closest_centroid = c;
        }
    }

    // set the closest cluster id for this datapoint/threadId
    cluster_assignment[id] = closest_centroid;
}

__global__ void resetCentroids(song *centroids){

    // get the id
    const int id = blockIdx.x * blockDim.x + threadIdx.x;
    // check bounds
    if (id >= K){return;}

    // reset the clusters to 0
    centroids[id].danceability = 0;
    centroids[id].energy =0;
    centroids[id].loudness = 0;
    centroids[id].speechiness = 0;
    centroids[id].acousticness = 0;
    centroids[id].instrumental = 0;
    centroids[id].liveness = 0;
    centroids[id].valence = 0;
    centroids[id].tempo = 0;
}

__global__ void sumCentroids(song *data, int *cluster_assignment, song *centroids, int *cluster_sizes){

    // get the id
    const int id = blockIdx.x * blockDim.x + threadIdx.x;
    // check bounds
    if (id >= N){return;}
            
            int cluster_id = cluster_assignment[id];

            // Sum the centroids variables up
            atomicAdd(&centroids[cluster_id].danceability, data[id].danceability);
            atomicAdd(&centroids[cluster_id].energy, data[id].energy);
            atomicAdd(&centroids[cluster_id].loudness, data[id].loudness);
            atomicAdd(&centroids[cluster_id].speechiness, data[id].speechiness);
            atomicAdd(&centroids[cluster_id].acousticness, data[id].acousticness);
            atomicAdd(&centroids[cluster_id].instrumental, data[id].instrumental);
            atomicAdd(&centroids[cluster_id].liveness, data[id].liveness);
            atomicAdd(&centroids[cluster_id].valence, data[id].valence);
            atomicAdd(&centroids[cluster_id].tempo, data[id].tempo);
            atomicAdd(&cluster_sizes[cluster_id], 1);
} 
__global__ void updateCentroids(song *data, int *cluster_assignment, song *centroids, int *cluster_sizes)
{
    // get the id
    const int id = blockIdx.x * blockDim.x + threadIdx.x;

    // check bounds
    if (id >= K){return;}

    // divide sums by the size
    centroids[id].danceability /= cluster_sizes[id];
    centroids[id].energy /= cluster_sizes[id];
    centroids[id].loudness /= cluster_sizes[id];
    centroids[id].speechiness /= cluster_sizes[id];
    centroids[id].acousticness /= cluster_sizes[id];
    centroids[id].instrumental /= cluster_sizes[id];
    centroids[id].liveness /= cluster_sizes[id];
    centroids[id].valence /= cluster_sizes[id];
    centroids[id].tempo /= cluster_sizes[id];
}

// int main()
void launcher(song *centroids_h, song *data_h, int *cluster_assignment_h)
{
    song *data_d;
    int *cluster_assignment_d;
    song *centroids_d;
    int *cluster_sizes_h = (int *)malloc(K * sizeof(int));
    int *cluster_sizes_d;
    dim3 block(std::ceil(N/64),1,1);
    dim3 grid(64,1,1);

    hipMalloc(&data_d, N * sizeof(song));
    hipMalloc(&cluster_assignment_d, N * sizeof(int));
    hipMalloc(&centroids_d, K * sizeof(song));
    hipMalloc(&cluster_sizes_d, K * sizeof(int));


    hipMemset(cluster_assignment_d, 0, N * sizeof(int));
    hipMemset(cluster_sizes_d, 0, K * sizeof(int));

    hipMemcpy(centroids_d, centroids_h, K * sizeof(song), hipMemcpyHostToDevice);
    hipMemcpy(data_d, data_h, N * sizeof(song), hipMemcpyHostToDevice);


    for(int i = 0; i < ROUNDS; i++)
    {
        findClosestCentroid<<<block, grid>>>(data_d, cluster_assignment_d, centroids_d);
        resetCentroids<<<block, grid>>>(centroids_d);
        hipDeviceSynchronize();
        hipMemset(cluster_sizes_d, 0, K * sizeof(int));
        sumCentroids<<<block, grid>>>(data_d, cluster_assignment_d, centroids_d, cluster_sizes_d);
        hipDeviceSynchronize();
        updateCentroids<<<block, grid>>>(data_d, cluster_assignment_d, centroids_d, cluster_sizes_d);
        hipDeviceSynchronize();

        // for debugging purposes
        hipMemcpy(centroids_h, centroids_d, K * sizeof(song), hipMemcpyDeviceToHost);
        for (int j = 0; j < K; ++j)
          {printf("Iteration %d: centroid %d: %f\n",i,j,centroids_h[j].danceability);}
    }

    // copy our final results over
    hipMemcpy(data_h, data_d, N * sizeof(song), hipMemcpyDeviceToHost);
    hipMemcpy(cluster_assignment_h, cluster_assignment_d, N * sizeof(int), hipMemcpyDeviceToHost);

    // free all of our memory
    hipFree(data_d);
    hipFree(cluster_assignment_d);
    hipFree(centroids_d);
    hipFree(cluster_sizes_d);
    free(cluster_sizes_h);
}

int main()
{
    int cluster_assignment_h[N * sizeof(int)];


    auto data = getCSV();
    auto centroids = generateCentroids(K, data);

    song *data_h = &data[0];
    song *centroids_h = &centroids[0];

    launcher(centroids_h, data_h, cluster_assignment_h);


    std::ofstream output_file("cudaResults.csv");
    output_file << "centroid,danceability,energy,loudness,speechiness,\
    acousticness,instrumental,liveness,valence,tempo\n";
    for (long unsigned int i = 0; i < N; ++i)
    {
        output_file << cluster_assignment_h[i] << "," << data_h[i].toString();
    }

    output_file.close();

    return 0;
}
