#include "hip/hip_runtime.h"
#include "cudaUtils.cuh"

extern "C++" void cudaMallocate(void **devPtr, size_t size)
{
    hipMalloc(devPtr, size);
}

extern "C++" void cudaMemorySet(void *devPtr, int value, size_t count)
{
    hipMemset(devPtr, value, count);
}

extern "C++" void cudaMemoryCopy(void *dst, const void *src, size_t count, int kind)
{
    hipMemcpy(dst, src, count, kind == 0 ? hipMemcpyHostToDevice : hipMemcpyDeviceToHost);
}

extern "C++" void findClosestCentroidExterior(song *data, int *cluster_assignment, song *centroids, int numSongs, int K, int blockDim1, int blockDim2, int blockDim3)
{
    dim3 block(std::ceil(numSongs/blockDim1),blockDim2,blockDim3);
    dim3 grid(blockDim1,blockDim2,blockDim3);
    findClosestCentroid<<<grid, block>>>(data, cluster_assignment, centroids, numSongs, K);
}

extern "C++" void resetCentroidsExterior(song *centroids, int K, int numSongs, int blockDim1, int blockDim2, int blockDim3)
{
    dim3 block(std::ceil(numSongs/blockDim1),blockDim2,blockDim3);
    dim3 grid(blockDim1,blockDim2,blockDim3);
    resetCentroids<<<grid, block>>>(centroids, K);
}

extern "C++" void cudaDeviceSync()
{
    hipDeviceSynchronize();
}

extern "C++" void sumCentroidsExterior(song *data, int *cluster_assignment, song *centroids, int *cluster_sizes, int numSongs, int blockDim1, int blockDim2, int blockDim3){
    dim3 block(std::ceil(numSongs/blockDim1),blockDim2,blockDim3);
    dim3 grid(blockDim1,blockDim2,blockDim3);
    sumCentroids<<<block, grid>>>(data, cluster_assignment, centroids, cluster_sizes, numSongs);
}

extern "C++" void updateCentroidsExterior(song *data, int *cluster_assignment, song *centroids, int *cluster_sizes, int K, int numSongs, int blockDim1, int blockDim2, int blockDim3)
{
    dim3 block(std::ceil(numSongs/blockDim1),blockDim2,blockDim3);
    dim3 grid(blockDim1,blockDim2,blockDim3);
    updateCentroids<<<block, grid>>>(data, cluster_assignment, centroids, cluster_sizes, K);
}

